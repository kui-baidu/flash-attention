#include "hip/hip_runtime.h"
#include "utils.h"

void SetZero(void *ptr, size_t sizeof_type, std::initializer_list<int> shapes, hipStream_t stream) {
    size_t n = sizeof_type;
    for (int s : shapes) n *= s;
    hipMemsetAsync(ptr, 0, n, stream);
}

template <typename T>
static __global__ void FillConstantKernel(T *ptr, T value, size_t n) {
  auto idx = static_cast<size_t>(blockDim.x) * blockIdx.x + threadIdx.x;
  if (idx < n) {
    ptr[idx] = value;
  }
} 

template <typename T>
void SetConstValue(void *ptr, T value, size_t n, hipStream_t stream) {
  constexpr auto kNumThreads = 1024;
  auto block = (n + kNumThreads - 1) / kNumThreads; 
  FillConstantKernel<T><<<block, kNumThreads, 0, stream>>>(static_cast<T *>(ptr), value, n);
} 

template
void SetConstValue(void *ptr, float value, size_t n, hipStream_t stream);
