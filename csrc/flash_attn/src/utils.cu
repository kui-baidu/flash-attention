#include "hip/hip_runtime.h"
#include "utils.h"

void SetZero(void *ptr, size_t sizeof_type, std::initializer_list<int> shapes, hipStream_t stream) {
    size_t n = sizeof_type;
    for (int s : shapes) n *= s;
    hipMemsetAsync(ptr, 0, n, stream);
}

static __global__ void _float2half(float *float_ptr, __half *half_ptr) {
    const int idx = threadIdx.x + blockDim.x * blockIdx.x;
    half_ptr[idx] = __float2half(float_ptr[idx]);
}

void Float2Half(void *float_ptr, void *half_ptr, hipStream_t stream) {
  constexpr auto kNumThreads = 1024;
  auto block = 512;
  _float2half<<<block, kNumThreads, 0, stream>>>(static_cast<float *>(float_ptr), static_cast<__half *>(float_ptr));
} 

template <typename T>
static __global__ void FillConstantKernel(T *ptr, T value, size_t n) {
  auto idx = static_cast<size_t>(blockDim.x) * blockIdx.x + threadIdx.x;
  if (idx < n) {
    ptr[idx] = value;
  }
} 

template <typename T>
void SetConstValue(void *ptr, T value, size_t n, hipStream_t stream) {
  constexpr auto kNumThreads = 1024;
  auto block = (n + kNumThreads - 1) / kNumThreads; 
  FillConstantKernel<T><<<block, kNumThreads, 0, stream>>>(static_cast<T *>(ptr), value, n);
} 

template
void SetConstValue(void *ptr, float value, size_t n, hipStream_t stream);
